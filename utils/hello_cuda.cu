// build nvcc -o hello_cuda  ./hello_cuda.cu
// run .\hello_cuda.exe

#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void helloCuda()
{
    printf("Hello CUDA world \n");
}

int main()
{
    int nx, ny;
    nx = 16;
    ny = 4;

    dim3 block(8, 2);
    dim3 grid(nx / block.x, ny / block.y);
    helloCuda<<<block, grid>>>();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}
