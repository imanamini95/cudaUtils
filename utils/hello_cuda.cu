// nvcc -o ./Release/output/hello_cuda ./utils/hello_cuda.cu
// ./Release/output/hello_cuda.exe

#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void helloCuda()
{
    printf("Hello CUDA world \n");
}

int main()
{
    int nx, ny;
    nx = 16;
    ny = 4;

    dim3 block(8, 2);
    dim3 grid(nx / block.x, ny / block.y);
    helloCuda<<<block, grid>>>();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}
