// nvcc -o ./Release/output/matrix_multiplication ./utils/matrix_multiplication.cu
// ./Release/output/matrix_multiplication.exe


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 3

__global__ void matrixMul(int *a, int *b, int *c)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (row < N && col < N)
    {
        for (int i = 0; i < N; i++)
            sum += a[row * N + i] * b[i * N + col];
        c[row * N + col] = sum;
    }
}

int main()
{
    int a[N][N], b[N][N], c[N][N];
    int *d_a, *d_b, *d_c;

    // Initialize matrices a and b
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            a[i][j] = i + j;
            b[i][j] = i - j;
        }
    }

    // Allocate device memory
    hipMalloc((void **)&d_a, N * N * sizeof(int));
    hipMalloc((void **)&d_b, N * N * sizeof(int));
    hipMalloc((void **)&d_c, N * N * sizeof(int));

    // Copy data to device memory
    hipMemcpy(d_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 grid(1, 1);
    dim3 block(N, N);

    // Launch kernel
    matrixMul<<<grid, block>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Display result
    printf("Matrix A:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d\t", a[i][j]);
        }
        printf("\n");
    }

    printf("\nMatrix B:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d\t", b[i][j]);
        }
        printf("\n");
    }

    printf("\nMatrix C:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d\t", c[i][j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
