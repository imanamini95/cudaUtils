#include "hip/hip_runtime.h"
// nvcc -o ./Release/output/device_query ./utils/device_query.cu
// ./Release/output/device_query.exe

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

void queryDevice()
{
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        printf("No CUDA supported device found");
    }

    int devNo = 0;
    hipDeviceProp_t iProp;
    hipGetDeviceProperties(&iProp, devNo);

    printf("Device %d: %s\n", devNo, iProp.name);
    printf("  Number of MPs: %d\n", iProp.multiProcessorCount);
    printf("  clock rate: %d\n", iProp.clockRate);
    printf("  compute capabilities: %d.%d\n", iProp.minor, iProp.major);
    printf("  Total global memory: %4.2f KB\n", iProp.totalGlobalMem / 1024.0);
}

int main()
{
    queryDevice();
    return 0;
}
