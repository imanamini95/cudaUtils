#include "hip/hip_runtime.h"
// nvcc -G -o ./Release/output/warp_divergence ./utils/warp_divergence.cu
// nvprof --metrics branch_efficiency ./Release/output/warp_divergence.exe

// experiment only works for GPU capability < 7.5 for higher Nsight

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_common.cuh"

__global__ void codeWithoutDivergence()
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    float a, b;
    a = b = 0;

    int warp_id = gid / 32;

    if (warp_id % 2 == 0)
    {
        a = 100.0;
        b = 50.0;
    }
    else
    {
        a = 200;
        b = 75;
    }
}

__global__ void divergenceCode()
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    float a, b;
    a = b = 0;

    if (gid % 2 == 0)
    {
        a = 100.0;
        b = 50.0;
    }
    else
    {
        a = 200;
        b = 75;
    }
}

int main(int argc, char **argv)
{
    printf("\n-----------------------WARP DIVERGENCE EXAMPLE------------------------ \n\n");

    int size = 1 << 22;

    dim3 block_size(128);
    dim3 grid_size((size + block_size.x - 1) / block_size.x);

    codeWithoutDivergence<<<grid_size, block_size>>>();
    hipDeviceSynchronize();

    divergenceCode<<<grid_size, block_size>>>();
    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}
